#include "hip/hip_runtime.h"
// Importing necessary header files
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"
#include <stdlib.h>
#include "moving_sphere.h"
#include "box.h"
#include <fstream>


// Error Handling in CUDA
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}


__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	vec3 cur_color = vec3(0.0, 0.0, 0.0);
	for (int i = 0; i < 50; i++) {
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
			vec3 attenuation;
			vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
				cur_attenuation *= attenuation;
				cur_attenuation = emitted + cur_attenuation;
				cur_ray = scattered;
			}
			else {
				//cur_attenuation *= attenuation;
				//cur_attenuation = emitted + cur_attenuation;
				return cur_attenuation;
			}
		}
		else {
			return vec3(0.0, 0.0, 0.0);
			return cur_attenuation * vec3(0.005, 0.005, 0.005);
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y() + 1.0f);
			vec3 c = (1.0f - t)*vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);
	for (int s = 0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r = (*cam)->get_ray(u, v, &local_rand_state);
		col += color(r, world, &local_rand_state);
	}
	rand_state[pixel_index] = local_rand_state;
	col /= float(ns);
	float f = 0.3; //Gamma correction
	col[0] = 1.85 * sqrt(col[0]) / f;
	col[1] = 1.85 * sqrt(col[1]) / f;
	col[2] = 1.85 * sqrt(col[2]) / f;
	if (col[0] > 1) {
		col[0] = 1;
	}
	if (col[1] > 1) {
		col[1] = 1;
	}
	if (col[2] > 1) {
		col[2] = 1;
	}

	fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))


// DEFINING DIFFERENT SCENES // SCENE 0,1,2

// EQUAL WORKLOAD
// SCENE DESCRIPTION: A room with 2 entities, one cuboid and other cube
__device__ void scene_0(hitable **HIT_LIST, hitable **HIT_WORLD, hiprandState *rand_state) {
	int i = 0;
	material *red = new lambertian(new constant_texture(vec3(0.65, 0.05, 0.05)));
	material *white = new lambertian(new constant_texture(vec3(0.73, 0.73, 0.73)));
	material *green = new lambertian(new constant_texture(vec3(0.12, 0.45, 0.15)));
	material *light = new diffuse_light_val(new constant_texture(vec3(15, 15, 15)));
	HIT_LIST[i++] = new flip_normals(new yz_rect(0, 555, 0, 555, 555, green));
	HIT_LIST[i++] = new yz_rect(0, 555, 0, 555, 0, red);
	HIT_LIST[i++] = new xz_rect(213, 343, 227, 332, 554, light);
	HIT_LIST[i++] = new flip_normals(new xz_rect(0, 555, 0, 555, 555, white));
	HIT_LIST[i++] = new xz_rect(0, 555, 0, 555, 0, white);
	HIT_LIST[i++] = new flip_normals(new xy_rect(0, 555, 0, 555, 555, white));
	HIT_LIST[i++] = new translate(
		new rotate_y(new box(vec3(0, 0, 0), vec3(165, 165, 165), white), -18),
		vec3(130, 0, 65)
	);
	HIT_LIST[i++] = new translate(
		new rotate_y(new box(vec3(0, 0, 0), vec3(165, 330, 165), white), 15),
		vec3(265, 0, 295)
	);
	*HIT_WORLD = new hitable_list(HIT_LIST, i);
}

// UNEQUAL WORKLOAD
// SCENE DESCIPTION: THREE SPHERES
__device__ void scene_1(hitable **HIT_LIST, hitable **HIT_WORLD, hiprandState *rand_state) {
	texture_val *checker = new checker_texture(
		new constant_texture(vec3(0.2, 0.3, 0.1)),
		new constant_texture(vec3(0.9, 0.9, 0.9))
	);
	hiprandState local_rand_state = *rand_state;
	*rand_state = local_rand_state;
	HIT_LIST[0] = new sphere(vec3(0, -1000, 0), 1000, new lambertian(checker));
	int i = 1;
	
	HIT_LIST[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
	HIT_LIST[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(new constant_texture(vec3(0.4, 0.2, 0.1))));
	HIT_LIST[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
	material *light = new diffuse_light_val(new constant_texture(vec3(15, 15, 15)));
	HIT_LIST[i++] = new xz_rect(213, 343, 227, 332, 554, light);
	*HIT_WORLD = new hitable_list(HIT_LIST, i);
}

// EXTREMELY UNEQUAL WORKLOAD
// SCENE DESCRIPTION: MULTIPLE SPHERES
__device__ void scene_2(hitable **HIT_LIST, hitable **HIT_WORLD, hiprandState *rand_state) {
	texture_val *checker = new checker_texture(
		new constant_texture(vec3(0.2, 0.3, 0.1)),
		new constant_texture(vec3(0.9, 0.9, 0.9))
	);
	hiprandState local_rand_state = *rand_state;
	*rand_state = local_rand_state;
	HIT_LIST[0] = new sphere(vec3(0, -1000, 0), 1000, new lambertian(checker));
	int i = 1;
	for (int a = -10; a < 10; a++) {
		for (int b = -10; b < 10; b++) {
			float choose_mat = RND;
			vec3 center(a + 0.9*RND, 0.2, b + 0.9*RND);
			if ((center - vec3(4, 0.2, 0)).length() > 0.9) {
				if (choose_mat < 0.8f) {
					HIT_LIST[i++] = new moving_sphere(
						center,
						center + vec3(0, 0.5*RND, 0),
						0.0, 1.0, 0.2,
						new lambertian(new constant_texture(
							vec3(RND*RND,
								RND*RND,
								RND*RND))));
				}
				else if (choose_mat < 0.95f) {
					HIT_LIST[i++] = new sphere(center, 0.2,
						new metal(vec3(0.5f*(1.0f + RND), 0.5f*(1.0f + RND), 0.5f*(1.0f + RND)), 0.5f*RND));
				}
				else {
					HIT_LIST[i++] = new sphere(center, 0.2, new dielectric(1.5));
				}
			}
		}
	}
	HIT_LIST[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
	HIT_LIST[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(new constant_texture(vec3(0.4, 0.2, 0.1))));
	HIT_LIST[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
	material *light = new diffuse_light_val(new constant_texture(vec3(15, 15, 15)));
	HIT_LIST[i++] = new xz_rect(213, 343, 227, 332, 554, light);
	*HIT_WORLD = new hitable_list(HIT_LIST, i);
}



// CREATING THE WORLD 
// -> INTIALIZING THE SCENE TO SET UP
// -> PLACING THE CAMERA

// -> PLEASE COMMENT THE SCENES WHICH ARE NOT BEING RENDERED AND ALSO THE CAMERA PLACEMENT CODE FOR THE CORRESPOINING SCENE

__global__ void create_world(hitable **HIT_LIST, hitable **HIT_WORLD, camera **CAMERA_LIST, int nx, int ny, hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprandState local_rand_state = *rand_state;
		*rand_state = local_rand_state;


		// SCENE 0
		// scene_0(HIT_LIST, HIT_WORLD, rand_state);
		// SCENE 1
		// scene_1(HIT_LIST, HIT_WORLD, rand_state);
		// SCENE 2
		scene_2(HIT_LIST, HIT_WORLD, rand_state);
		

		// CAMERA PLACEMENT FOR SCENE 0


		// vec3 lookfrom(278, 278, -800);
		// vec3 lookat(278, 278, 0);
		// float dist_to_focus = 10.0; (lookfrom - lookat).length();
		// float aperture = 0.0;
		// float vfov = 40;


		// CAMERA PLACEMENT FOR SCENE 1 & SCENE 2
		
		vec3 lookfrom(13, 2, 3);
		vec3 lookat(0, 0, 0);
		float dist_to_focus = 10.0; (lookfrom - lookat).length();
		float aperture = 0.0;
		float vfov = 50;
		


		*CAMERA_LIST = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			vfov,
			float(nx) / float(ny),
			aperture,
			dist_to_focus,
			0.0,1.0);
	}
}

// FREEING UP THE MEMORY FOR THE CREATION OF THE ENVIRONMENT
__global__ void free_world(hitable **HIT_LIST, hitable **HIT_WORLD, camera **CAMERA_LIST) {
	for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
		delete ((sphere *)HIT_LIST[i])->mat_ptr;
		delete HIT_LIST[i];
	}
	delete *HIT_WORLD;
	delete *CAMERA_LIST;
}

int main() {
	// TX-> DEFINING THE THREAD SIZE
	int tx = 64; // 32 * 32 	// THREAD X
	int ty = 8;	 		// THREAD Y

	// height and width of an image
	int width = 512; //256
	int height = 512; //256
	int num_samples = 128; //8
	
	std::ofstream output_file;
	output_file.open("output.ppm");
	std::cerr << "Rendering a " << width << "x" << height << " image with " << num_samples << " samples per pixel ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = width * height;	// NUMBER OF PIXELS-> SIZE OF THE RESOLUTION
	size_t fb_size = num_pixels * sizeof(vec3); // DEFINING THE FRAME BUFFER


	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	hiprandState *d_rand_state;
	checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
	hiprandState *d_rand_state2;
	checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1 * sizeof(hiprandState)));

	// we need that 2nd random state to be initialized for the world creation
	rand_init << <1, 1 >> > (d_rand_state2);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());


	// DEFINE THE HITABLES AND CAMERA LIST 

	hitable **HIT_LIST;

	int num_hitables = 22 * 22 + 1 + 3;
	checkCudaErrors(hipMalloc((void **)&HIT_LIST, num_hitables * sizeof(hitable *)));

	hitable **HIT_WORLD;
	checkCudaErrors(hipMalloc((void **)&HIT_WORLD, sizeof(hitable *)));

	camera **CAMERA_LIST;
	checkCudaErrors(hipMalloc((void **)&CAMERA_LIST, sizeof(camera *)));

	create_world << <1, 1 >> > (HIT_LIST, HIT_WORLD, CAMERA_LIST, width, height, d_rand_state2);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	clock_t start, stop;
	std::cerr << "STARTING THE RENDERER ................\n";
	start = clock();
	// DEFINING THE BLOCKS DIMENSIONS
	dim3 blocks(width / tx + 1, height / ty + 1);
	// DEFINING THE THREAD DIMENSIONS
	dim3 threads(tx, ty);

	// RENDERING PROCES....
	render_init << <blocks, threads >> > (width, height, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	render << <blocks, threads >> > (fb, width, height, num_samples, CAMERA_LIST, HIT_WORLD, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	stop = clock();

	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";


	// RENDERING THE FINAL OUTPUT AS IMAGE
	output_file << "P3\n" << width << " " << height << "\n255\n";
	for (int j = height - 1; j >= 0; j--) {

		for (int i = 0; i < width; i++) {

			size_t pixel_index = j * width + i;
			// red pixels
			int INT_R = int(255.99*fb[pixel_index].r());
			// green pixels
			int INT_G = int(255.99*fb[pixel_index].g());
			// blue pixels
			int INT_B = int(255.99*fb[pixel_index].b());

			output_file << INT_R << " " << INT_G << " " << INT_B << "\n";
		}
	}

	output_file.close();

	// clean up

	checkCudaErrors(hipDeviceSynchronize());

	free_world << <1, 1 >> > (HIT_LIST, HIT_WORLD, CAMERA_LIST);

	checkCudaErrors(hipGetLastError());
	// checkCudaErrors(hipFree(CAMERA_LIST));
	checkCudaErrors(hipFree(HIT_WORLD));
	checkCudaErrors(hipFree(HIT_LIST));
	checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipFree(fb));

	hipDeviceReset();
}